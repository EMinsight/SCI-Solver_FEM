#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include "FEMSolver.h"
#include <cuda_resources.h>

/**
 * Runs an "egg carton example to test accuracy"
 */
int main(int argc, char** argv)
{
  FEMSolver cfg;
  cfg.filename_ = "CubeMesh_size256step16_correct";
  cfg.verbose_ = true;
  if (argc > 1) {
    cfg.filename_ = std::string(argv[1]);
  }
  cfg.tetMesh_ = TetMesh::read(
    (cfg.filename_ + ".node").c_str(),
    (cfg.filename_ + ".ele").c_str(), true, cfg.verbose_);
  size_t num_vert = cfg.tetMesh_->vertices.size();
  float lambda = 1.f;
  //create the A matrix
  Matrix_ell_h A_h(num_vert, num_vert, num_vert, 1);
  for (int i = 0; i < num_vert; i++) {
    A_h.column_indices(i, 0) = i;
    A_h.values(i, 0) = 8. * M_PI * M_PI + lambda;
  }
  //create the b vector
  Vector_h_CG b_h(num_vert, 1.0), x_h(num_vert, 0.0);
  for (int i = 0; i < num_vert; i++) {
    b_h[i] = 8. * M_PI * M_PI * std::sin(2. * M_PI * cfg.tetMesh_->vertices[i][0]) *
      std::sin(2. * M_PI * cfg.tetMesh_->vertices[i][1]) + lambda *
      std::sin(2. * M_PI * cfg.tetMesh_->vertices[i][0]) *
      std::sin(2. * M_PI * cfg.tetMesh_->vertices[i][1]);
  }
  //create the b vector
  std::vector<float> x_answer;
  for (int i = 0; i < num_vert; i++) {
    x_answer.push_back(
      std::sin(2. * M_PI * cfg.tetMesh_->vertices[i][0]) *
      std::sin(2. * M_PI * cfg.tetMesh_->vertices[i][1]));
  }
  cfg.checkMatrixForValidContents(&A_h);
  cfg.solveFEM(&A_h, &x_h, &b_h);
  //look for error
  float error = 0.f;
  std::vector<float> x_actual;
  for (int i = 0; i < num_vert; i++) {
    error += (x_h[i] - x_answer[i]) * (x_h[i] - x_answer[i]);
    x_actual.push_back(x_h[i]);
  }
  cfg.writeVTK(x_actual, "test_egg_carton");
}
