#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include "setup_solver.h"
#include "cuda_resources.h"

/**
 * SCI-Solver_FEM :: Example 3
 * This example is the basic steps for running the solver:
 *  1. We define our main AMG_Config object.
 *  2. We set all of the parameters we want. (Otherwise defaults used.)
 *  3. We read in our input data mesh.
 *  4. We declare all the variables we need for the solver (matrices).
 *  5. We invoke the "setup_solver" call, which does all of the work.
 */

void printElementWithHeader(vector<double>& test, unsigned int index)
{
  std::cout << "element #" << index << " = " << test[index] << std::endl;
}

void printMatlabReadContents(vector<double>& test)
{
  std::cout << "test result vector is size = " << test.size() << std::endl;
  int incr = test.size() / 5;
  for (int j = 0; j < test.size(); j += incr) {
    printElementWithHeader(test, j);
  }
  if( test.size() > 0 )
    std::cout << "last element = " << test[test.size() - 1] << std::endl;
}

int importRhsVectorFromFile(string filename, Vector_h_CG& targetVector, bool verbose)
{
  vector<double> sourceRead;

  if( filename.empty() ) {
    string errMsg = "No matlab file provided for RHS (b) vector.";
    errMsg += " Specify the file using argument at commandline using -b switch.";
    std::cerr << errMsg << std::endl;
    return -1;
  }
  if( readMatlabNormalMatrix(filename, &sourceRead) < 0 ) {
    std::cerr << "Failed to read matlab file for RSH (b)." << std::endl;
    return -1;
  }
  targetVector = sourceRead;
  if (verbose) {
    int sizeRead = targetVector.size();
    std::cout << "Finished reading RHS (b) data file with ";
    std::cout << sizeRead << " values." << std::endl;
  }
  return 0;
}

int importStiffnessMatrixFromFile(string filename, Matrix_ell_h* targetMatrix, bool verbose)
{
  if( filename.empty() ) {
    string errMsg = "No matlab file provided for stiffness matrix (A).";
    errMsg += " Specify the file using argument at commandline using -A switch.";
    std::cerr << errMsg << std::endl;
    return -1;
  }
  if( readMatlabSparseMatrix(filename, targetMatrix) != 0 ) {
    std::cerr << "Failed to read matlab file for stiffness matrix (A)." << std::endl;
    return -1;
  }
  if (verbose) {
    string msg = "Finished reading stiffness matrix.";
    std::cout << msg << std::endl;
  }
  return 0;
}

void debugPrintMatlabels(TetMesh* mesh)
{
  std::cout << "Found " << mesh->matlabels.size() << " elements in matlabels." << std::endl;
  unsigned int numZeros = 0;
  for (vector<int>::iterator it = mesh->matlabels.begin(); it != mesh->matlabels.end(); ++it)
  {
    if( (*it) == 0 )
      numZeros++;
    else
      std::cout << (*it) << std::endl;
  }
  std::cout << numZeros << " zero values found." << std::endl;
}


int main(int argc, char** argv)
{
  //Verbose option
  bool verbose = false;
  bool zero_based = false;
  std::string filename, aFilename, bFilename;
  for (int i = 0; i < argc; i++) {
    if (strcmp(argv[i],"-v") == 0) {
      verbose = true;
    } else if (strcmp(argv[i],"-i") == 0) {
      if (i+1 >= argc)
    	break;
      filename = std::string(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-b") == 0) {
      if (i+1 >= argc)
    	break;
      bFilename = std::string(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-A") == 0) {
      if (i+1 >= argc)
    	break;
      aFilename = std::string(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-z") == 0) {
      zero_based = true;
    }
  }
  //Our main configuration object. We will set aspects where the
  // default values are not what we desire.
  AMG_Config cfg;
  //assuming our device is zero...
  int dev_num = 0;
  cfg.setParameter("cuda_device_num", dev_num);
  // Make sure part_max_size is representative of harware limits by default
  // when compiling the library, up to 64 registers were seen to be used on the
  // device. We can set our max allocation based on that number
  int max_registers_used = 64;
  cfg.setParameter("part_max_size", getMaxThreads(max_registers_used,dev_num));
  //set the desired algorithm
  cfg.setParameter("algorithm", /*(AlgorithmType::)*/CLASSICAL);
  //set the convergence tolerance
  cfg.setParameter("tolerance", 1e-8);
  //set the weight parameter used in a smoother
  cfg.setParameter("smoother_weight", 0.7);
  //set the weight parameter used in a prolongator smoother
  cfg.setParameter("pro_omega", 0.7);
  //set the maximum solve iterations
  cfg.setParameter("max_iters", 10);
  //set the pre inner iterations for GSINNER
  cfg.setParameter("PreINNER_iters", 2);
  //set the post inner iterations for GSINNER
  cfg.setParameter("PostINNER_iters", 3);
  //set the Aggregator METIS (0) or MIS (1)
  cfg.setParameter("aggregator_type", 1);
  //set the Max size of coarsest level
  cfg.setParameter("metis_size", 90102);
  //set the solving algorithm
  cfg.setParameter("solver", /*(SolverType::)*/PCG_SOLVER);
  //set the cycle algorithm
  cfg.setParameter("cycle", /*(CycleType::)*/V_CYCLE);
  //set the convergence tolerance algorithm
  cfg.setParameter("convergence", /*(ConvergenceType::)*/ABSOLUTE_CONVERGENCE);
  //set the smoothing algorithm
  cfg.setParameter("smoother", /*(SmootherType::)*/GAUSSSEIDEL);
  //Now we read in the mesh of choice
  //TriMesh* meshPtr = TriMesh::read("mesh.ply"); //-----if we were reading a Triangle mesh

  //read in the Tetmesh
  if (filename.empty())
    filename = std::string("../example_data/CubeMesh_size256step16");
  if (verbose)
    std::cout << "Reading in file: " << filename << std::endl;
  TetMesh* tetmeshPtr = TetMesh::read(
      (filename + ".node").c_str(),
      (filename + ".ele").c_str(), zero_based, verbose);

  //The stiffness matrix A 
  Matrix_ell_h A_h;
  //get the basic stiffness matrix (constant) by creating the mesh matrix
  getMatrixFromMesh(cfg, tetmeshPtr, &A_h, true, verbose);

  //Import right-hand-side single-column array (b)
  Vector_h_CG b_h;
  if( importRhsVectorFromFile(bFilename, b_h, verbose) < 0 )
    return 0;

  //The answer vector.
  Vector_h_CG x_h(A_h.num_rows, 0.0); //intial X vector

#define USE_IDENTITY_MATRIX_ONLY

#ifndef USE_IDENTITY_MATRIX_ONLY
  //Import stiffness matrix (A)
  Matrix_ell_h A_h_import;
  if( importStiffnessMatrixFromFile(aFilename, &A_h_import, verbose) < 0 )
    return 0;
#else //USE_IDENTITY_MATRIX_ONLY
  Matrix_ell_h A_h_import(A_h.num_rows, A_h.num_rows, A_h.num_rows, 1.0);
  for (int i = 0; i < A_h.num_rows; i++) {
    A_h_import.column_indices(i, 0) = i;
    A_h_import.values(i, 0) = 1;
  }
#endif //#ifdef USE_IDENTITY_MATRIX_ONLY

  //multiply the mesh matrix by the stiffness properties matrix.
  Matrix_ell_h out;
  cusp::multiply(A_h_import, A_h, out);
  A_h = Matrix_ell_h(out);

  if( verbose )
    std::cout << "Calling setup_solver." << std::endl;
  //The final call to the solver
  checkMatrixForValidContents(&A_h, verbose);
  Matrix_ell_d A_d(A_h);

  setup_solver(cfg, tetmeshPtr, &A_d, &x_h, &b_h, verbose);
  //At this point, you can do what you need with the matrices.
  if (writeMatlabArray("output.mat", x_h)) {
    std::cerr << "failed to write matlab file." << std::endl;
  }
  return 0;
}
