#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int getMaxThreads(
    const int max_regs_per_thread, 
    int cuda_device) {
  hipDeviceProp_t d;
  hipGetDeviceProperties(&d, cuda_device);
  return d.regsPerBlock / max_regs_per_thread;
}
