#include "hip/hip_runtime.h"
#include <smoothedMG/aggregators/misHelpers.h>
extern "C" {
#include "metis.h"
}
#include "AggMIS_Types.h"
#include "AggMIS_Aggregation_GPU.h"
#include "AggMIS_MIS_GPU.h"
#include "AggMIS_MIS_CPU.h"
#include "AggMIS_MergeSplitConditioner.h"
#include "AggMIS_MergeSplitConditioner_CPU.h"
#include "AggMIS_GraphHelpers.h"
#include <smoothedMG/aggregators/Timer.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include "Logger.h"
namespace misHelpers {
   double totalAggregationTime = 0;
   int totalAggregationCalls = 0;
   namespace CP {
      void OldMIS(IdxVector_d &adjIndexesIn,
            IdxVector_d &adjacencyIn,
            IdxVector_d &permutation,
            IdxVector_d &ipermutation,
            IdxVector_d &aggregateIdx,
            IdxVector_d &partitionIdx,
            IdxVector_d &partitionLabel,
            IdxVector_d &adjIndexesOut,
            IdxVector_d &adjacencyOut,
            int parameters,
            int part_max_size,
            bool verbose) {

         int numNodesIn = adjIndexesIn.size() - 1; // Size of input graph
         int fineDepth = parameters % 100; // The MIS depth for the first aggregation
         int coarseDepth = (parameters / 100) % 100; // The MIS depth for the second aggregation
         int minAggregateSize = (parameters / 10000) % 10; // The minimum acceptable size for an aggregate

         IdxVector_d fineAggregate(numNodesIn, 0); // The partition label for the fine partition
         IdxVector_d fineAggregateSort; // The copy of the fine partition label that is sorted
         IdxVector_d finePartSizes; // Vector with sizes of fine partitions
         IdxVector_d neighborCountsIn; // Vector to hold the sizes for each nodes adjacency
         IdxVector_d permutedAdjIndexesIn; // Vector to hold the indices for the initial adjacency permutation
         IdxVector_d permutedAdjacencyIn; // Holds the permuted initial adjacency
         IdxVector_d neighborCountsOut; // Holds the counts of neighbors for the induced graph
         IdxVector_d coarseAggregate; // Holds the partition label for the coarse partition
         IdxVector_d aggregateRemapId; // Holds the current id of each aggregate
         IdxVector_d iAggregateRemapId; // Holds the permutation to remap the aggregate id'
         IdxVector_d aggregateRemapIndex; // Holds the start index of each aggregate
         IdxVector_d inducedNodeWeights; // Holds the sizes of the fine aggregates
         AggMIS::Types::JTimer timmy;
         if (verbose)
            std::cout << "Finished initializing IdxVector_d elements." << std::endl;

         partitionLabel = IdxVector_d(numNodesIn, 2); // Holds the partition each vertex is located in
         if (verbose)
            std::cout << "Finished with partitionLabel creation." << std::endl;
         timmy.start();

         misHelpers::aggregateGraph(minAggregateSize, fineDepth, adjIndexesIn, adjacencyIn, fineAggregate, verbose);
         if (verbose)
            std::cout << "Finished with aggregateGraph." << std::endl;
         timmy.stop();
         totalAggregationTime += timmy.getElapsedTimeInSec(true);
         if (verbose)
            printf("Fine conditioning time: %3.3fs\n", timmy.getElapsedTimeInSec(true));
         totalAggregationCalls++;

         Help::RecordAllStats(adjIndexesIn, adjacencyIn, fineAggregate, "Fine Aggregation");

         // Setting the permutation array to have values equal to element indices
         permutation = IdxVector_d(numNodesIn);
         misHelpers::fillWithIndex(permutation);

         // Sorting arrays together:
         fineAggregateSort = fineAggregate;
         thrust::sort_by_key(fineAggregateSort.begin(), fineAggregateSort.end(), permutation.begin());
         if (verbose)
            std::cout << "Finished with fineAggregateSort." << std::endl;

         // Building the permutation array:
         misHelpers::getInversePermutation(permutation, ipermutation);
         if (verbose)
            std::cout << "Got permutation array." << std::endl;

         // Getting the aggregate indices and node weights for the induced graph
         misHelpers::getPartSizes(fineAggregateSort, inducedNodeWeights, aggregateIdx);
         if (verbose)
            std::cout << "Got partition sizes." << std::endl;

         // Getting the induced graph:
         misHelpers::getInducedGraph(adjIndexesIn, adjacencyIn, fineAggregate, adjIndexesOut, adjacencyOut);
         if (verbose)
            std::cout << "Got induced graph." << std::endl;

         // Doing the coarse aggregation:
         int maxSize = part_max_size;
         int fullSize = adjIndexesIn.size() - 1;
         coarseAggregate = IdxVector_d(fullSize, 1);
         int inducedGraphSize = adjIndexesOut.size() - 1;

         timmy.start();
         misHelpers::aggregateWeightedGraph(maxSize, fullSize, coarseDepth, adjIndexesOut, adjacencyOut, coarseAggregate, inducedNodeWeights, verbose);
         timmy.stop();
         if (verbose)
            std::cout << "Finished aggregateWeightedGraph." << std::endl;

         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Coarse Aggregation");

         // Performing new version of getting induced graph
         misHelpers::remapInducedGraph(adjIndexesOut, adjacencyOut, coarseAggregate);
         if (verbose)
            std::cout << "Finished remapInducedGraph." << std::endl;

         // Filling in the partitionLabel:
         misHelpers::fillPartitionLabel(coarseAggregate, fineAggregateSort, partitionLabel);
         if (verbose)
            std::cout << "Finished fillPartitionLabel." << std::endl;

         // Do a stable sort by key with the partitionLabel as the key:
         thrust::stable_sort_by_key(partitionLabel.begin(), partitionLabel.end(), thrust::make_zip_iterator(thrust::make_tuple(fineAggregateSort.begin(), permutation.begin())));
         if (verbose)
            std::cout << "Finished thrust::stable_sort_by_key." << std::endl;

         // Remapping the aggregate id's:
         aggregateRemapId = IdxVector_d(aggregateIdx.size() - 1, 0);
         aggregateRemapIndex = IdxVector_d(aggregateIdx.size() - 1, 0);
         misHelpers::fillWithIndex(aggregateRemapId);
         misHelpers::getAggregateStartIndices(fineAggregateSort, aggregateRemapIndex);
         thrust::stable_sort_by_key(aggregateRemapIndex.begin(), aggregateRemapIndex.end(), aggregateRemapId.begin());
         misHelpers::getInversePermutation(aggregateRemapId, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregateSort, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregate, iAggregateRemapId);

         // Sort the coarseAggregate for indices and permutation:
         thrust::sort(coarseAggregate.begin(), coarseAggregate.end());
         misHelpers::getPartIndices(coarseAggregate, partitionIdx);

         // Get indices for the fine aggregates
         misHelpers::getPartIndices(fineAggregateSort, aggregateIdx);

         // Putting in the right permutation vectors for the output:
         ipermutation = permutation;
         misHelpers::getInversePermutation(ipermutation, permutation);
      }
      void MetisBottomUp(IdxVector_d &adjIndexesIn,
            IdxVector_d &adjacencyIn,
            IdxVector_d &permutation,
            IdxVector_d &ipermutation,
            IdxVector_d &aggregateIdx,
            IdxVector_d &partitionIdx,
            IdxVector_d &partitionLabel,
            IdxVector_d &adjIndexesOut,
            IdxVector_d &adjacencyOut,
            int parameters,
            int part_max_size,
            bool verbose) {
         int numNodesIn = adjIndexesIn.size() - 1; // Size of input graph

         IdxVector_d fineAggregate(numNodesIn, 0); // The partition label for the fine partition
         IdxVector_d fineAggregateSort; // The copy of the fine partition label that is sorted
         IdxVector_d finePartSizes; // Vector with sizes of fine partitions
         IdxVector_d neighborCountsIn; // Vector to hold the sizes for each nodes adjacency
         IdxVector_d permutedAdjIndexesIn; // Vector to hold the indices for the initial adjacency permutation
         IdxVector_d permutedAdjacencyIn; // Holds the permuted initial adjacency
         IdxVector_d neighborCountsOut; // Holds the counts of neighbors for the induced graph
         IdxVector_d coarseAggregate; // Holds the partition label for the coarse partition
         IdxVector_d aggregateRemapId; // Holds the current id of each aggregate
         IdxVector_d iAggregateRemapId; // Holds the permutation to remap the aggregate id'
         IdxVector_d aggregateRemapIndex; // Holds the start index of each aggregate
         IdxVector_d inducedNodeWeights; // Holds the sizes of the fine aggregates

         partitionLabel = IdxVector_d(numNodesIn, 2); // Holds the partition each vertex is located in
         int fineSize, coarseSize;
         coarseSize = part_max_size % 1000;
         fineSize = (part_max_size / 1000) % 1000;


         // Getting the fine aggregation with Metis
         AT::IntVector_h indices(adjIndexesIn.size());
         thrust::copy(adjIndexesIn.begin(), adjIndexesIn.end(), indices.begin());
         AT::IntVector_h adjacency(adjacencyIn.size());
         thrust::copy(adjacencyIn.begin(), adjacencyIn.end(), adjacency.begin());
         AT::IntVector_h result(numNodesIn);
         Help::GetMetisAggregation(indices, adjacency, result, fineSize);
         thrust::copy(result.begin(), result.end(), fineAggregate.begin());

         Help::RecordAllStats(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               "Fine Aggregation");

         // Setting the permutation array to have values equal to element indices
         permutation = IdxVector_d(numNodesIn);
         misHelpers::fillWithIndex(permutation);

         // Sorting arrays together:
         fineAggregateSort = fineAggregate;
         thrust::sort_by_key(fineAggregateSort.begin(), fineAggregateSort.end(), permutation.begin());

         // Building the permutation array:
         misHelpers::getInversePermutation(permutation, ipermutation);

         // Getting the aggregate indices and node weights for the induced graph
         misHelpers::getPartSizes(fineAggregateSort, inducedNodeWeights, aggregateIdx);

         // Getting the induced graph:
         misHelpers::getInducedGraph(adjIndexesIn, adjacencyIn, fineAggregate, adjIndexesOut, adjacencyOut);

         int inducedGraphSize = adjIndexesOut.size() - 1;

         // Doing the coarse aggregation (assuming Metis parts are close enough in size to ignore weighting)
         indices.assign(adjIndexesOut.begin(), adjIndexesOut.end());
         adjacency.assign(adjacencyOut.begin(), adjacencyOut.end());
         Help::GetMetisAggregation(indices, adjacency, result, coarseSize);
         coarseAggregate.assign(result.begin(), result.end());

         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Coarse Aggregation");

         // Performing new version of getting induced graph
         misHelpers::remapInducedGraph(adjIndexesOut, adjacencyOut, coarseAggregate);

         // Filling in the partitionLabel:
         misHelpers::fillPartitionLabel(coarseAggregate, fineAggregateSort, partitionLabel);

         // Do a stable sort by key with the partitionLabel as the key:
         thrust::stable_sort_by_key(partitionLabel.begin(), partitionLabel.end(), thrust::make_zip_iterator(thrust::make_tuple(fineAggregateSort.begin(), permutation.begin())));

         // Remapping the aggregate id's:
         aggregateRemapId = IdxVector_d(aggregateIdx.size() - 1, 0);
         aggregateRemapIndex = IdxVector_d(aggregateIdx.size() - 1, 0);
         misHelpers::fillWithIndex(aggregateRemapId);
         misHelpers::getAggregateStartIndices(fineAggregateSort, aggregateRemapIndex);
         thrust::stable_sort_by_key(aggregateRemapIndex.begin(), aggregateRemapIndex.end(), aggregateRemapId.begin());
         misHelpers::getInversePermutation(aggregateRemapId, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregateSort, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregate, iAggregateRemapId);

         // Sort the coarseAggregate for indices and permutation:
         thrust::sort(coarseAggregate.begin(), coarseAggregate.end());
         misHelpers::getPartIndices(coarseAggregate, partitionIdx);

         // Get indices for the fine aggregates
         misHelpers::getPartIndices(fineAggregateSort, aggregateIdx);

         // Putting in the right permutation vectors for the output:
         ipermutation = permutation;
         misHelpers::getInversePermutation(ipermutation, permutation);

         // Clean up temp vectors
         indices.clear();
         adjacency.clear();
         result.clear();
         if (verbose)
            printf("Total aggregation time (Metis): %3.4fs for %d calls\n", totalAggregationTime, totalAggregationCalls);
      }
      void MetisTopDown(IdxVector_d &adjIndexesIn,
            IdxVector_d &adjacencyIn,
            IdxVector_d &permutation,
            IdxVector_d &ipermutation,
            IdxVector_d &aggregateIdx,
            IdxVector_d &partitionIdx,
            IdxVector_d &partitionLabel,
            IdxVector_d &adjIndexesOut,
            IdxVector_d &adjacencyOut,
            int parameters,
            int part_max_size,
            bool verbose) {
         int numNodesIn = adjIndexesIn.size() - 1; // Size of input graph
         int fineDepth = parameters % 100; // The MIS depth for the first aggregation
         int coarseDepth = (parameters / 100) % 100; // The MIS depth for the second aggregation
         int minAggregateSize = (parameters / 10000) % 10; // The minimum acceptable size for an aggregate
         IdxVector_d fineAggregate(numNodesIn, 0); // The partition label for the fine partition
         IdxVector_d fineAggregateSort; // The copy of the fine partition label that is sorted
         IdxVector_d finePartSizes; // Vector with sizes of fine partitions
         IdxVector_d neighborCountsIn; // Vector to hold the sizes for each nodes adjacency
         IdxVector_d permutedAdjIndexesIn; // Vector to hold the indices for the initial adjacency permutation
         IdxVector_d permutedAdjacencyIn; // Holds the permuted initial adjacency
         IdxVector_d neighborCountsOut; // Holds the counts of neighbors for the induced graph
         IdxVector_d coarseAggregate; // Holds the partition label for the coarse partition
         IdxVector_d aggregateRemapId; // Holds the current id of each aggregate
         IdxVector_d iAggregateRemapId; // Holds the permutation to remap the aggregate id'
         IdxVector_d aggregateRemapIndex; // Holds the start index of each aggregate
         IdxVector_d inducedNodeWeights; // Holds the sizes of the fine aggregates

         partitionLabel = IdxVector_d(numNodesIn, 2); // Holds the partition each vertex is located in
         misHelpers::aggregateGraph(minAggregateSize, fineDepth, adjIndexesIn, adjacencyIn, fineAggregate, verbose);

         // Setting the permutation array to have values equal to element indices
         permutation = IdxVector_d(numNodesIn);
         misHelpers::fillWithIndex(permutation);

         // Sorting arrays together:
         fineAggregateSort = fineAggregate;
         thrust::sort_by_key(fineAggregateSort.begin(), fineAggregateSort.end(), permutation.begin());

         // Building the permutation array:
         misHelpers::getInversePermutation(permutation, ipermutation);

         // Getting the aggregate indices and node weights for the induced graph
         misHelpers::getPartSizes(fineAggregateSort, inducedNodeWeights, aggregateIdx);

         // Getting the induced graph:
         misHelpers::getInducedGraph(adjIndexesIn, adjacencyIn, fineAggregate, adjIndexesOut, adjacencyOut);

         // Doing the coarse aggregation:
         int maxSize = part_max_size; //400;
         int fullSize = adjIndexesIn.size() - 1;
         coarseAggregate = IdxVector_d(fullSize, 1);
         misHelpers::aggregateWeightedGraph(maxSize, fullSize, coarseDepth, adjIndexesOut, adjacencyOut, coarseAggregate, inducedNodeWeights, verbose);

         // Performing new version of getting induced graph
         misHelpers::remapInducedGraph(adjIndexesOut, adjacencyOut, coarseAggregate);

         // Filling in the partitionLabel:
         misHelpers::fillPartitionLabel(coarseAggregate, fineAggregateSort, partitionLabel);

         // Do a stable sort by key with the partitionLabel as the key:
         thrust::stable_sort_by_key(partitionLabel.begin(), partitionLabel.end(), thrust::make_zip_iterator(thrust::make_tuple(fineAggregateSort.begin(), permutation.begin())));

         // Remapping the aggregate id's:
         aggregateRemapId = IdxVector_d(aggregateIdx.size() - 1, 0);
         aggregateRemapIndex = IdxVector_d(aggregateIdx.size() - 1, 0);
         misHelpers::fillWithIndex(aggregateRemapId);
         misHelpers::getAggregateStartIndices(fineAggregateSort, aggregateRemapIndex);
         thrust::stable_sort_by_key(aggregateRemapIndex.begin(), aggregateRemapIndex.end(), aggregateRemapId.begin());
         misHelpers::getInversePermutation(aggregateRemapId, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregateSort, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregate, iAggregateRemapId);

         // Sort the coarseAggregate for indices and permutation:
         thrust::sort(coarseAggregate.begin(), coarseAggregate.end());
         misHelpers::getPartIndices(coarseAggregate, partitionIdx);

         // Get indices for the fine aggregates
         misHelpers::getPartIndices(fineAggregateSort, aggregateIdx);

         // Putting in the right permutation vectors for the output:
         ipermutation = permutation;
         misHelpers::getInversePermutation(ipermutation, permutation);
      }
      void NewMIS(IdxVector_d &adjIndexesIn,
            IdxVector_d &adjacencyIn,
            IdxVector_d &permutation,
            IdxVector_d &ipermutation,
            IdxVector_d &aggregateIdx,
            IdxVector_d &partitionIdx,
            IdxVector_d &partitionLabel,
            IdxVector_d &adjIndexesOut,
            IdxVector_d &adjacencyOut,
            int parameters,
            int part_max_size,
            bool verbose) {
         int numNodesIn = adjIndexesIn.size() - 1; // Size of input graph
         int fineDepth = parameters % 100; // The MIS depth for the first aggregation
         int coarseDepth = (parameters / 100) % 100;
         IdxVector_d fineAggregate(numNodesIn, 0); // The partition label for the fine partition
         IdxVector_d fineAggregateSort; // The copy of the fine partition label that is sorted
         IdxVector_d finePartSizes; // Vector with sizes of fine partitions
         IdxVector_d neighborCountsIn; // Vector to hold the sizes for each nodes adjacency
         IdxVector_d permutedAdjIndexesIn; // Vector to hold the indices for the initial adjacency permutation
         IdxVector_d permutedAdjacencyIn; // Holds the permuted initial adjacency
         IdxVector_d neighborCountsOut; // Holds the counts of neighbors for the induced graph
         IdxVector_d coarseAggregate; // Holds the partition label for the coarse partition
         IdxVector_d aggregateRemapId; // Holds the current id of each aggregate
         IdxVector_d iAggregateRemapId; // Holds the permutation to remap the aggregate id'
         IdxVector_d aggregateRemapIndex; // Holds the start index of each aggregate
         IdxVector_d inducedNodeWeights; // Holds the sizes of the fine aggregates

         AggMIS::Types::JTimer jimmy;
         AggMIS::Types::JTimer iTime;
         int fineMin, fineMax, coarseMin, coarseMax;
         fineMax = parameters % 1000;
         fineMin = (parameters / 1000) % 1000;
         coarseMax = part_max_size % 1000;
         coarseMin = (part_max_size / 1000) % 1000;
         coarseDepth = (parameters / 1000000) % 10;
         fineDepth = (parameters / 10000000) % 10;


         partitionLabel = IdxVector_d(numNodesIn, 2); // Holds the partition each vertex is located in

         // First transfer in the graph
         AT::Graph_d fineGraph;
         fineGraph.indices->swap(adjIndexesIn);
         fineGraph.adjacency->swap(adjacencyIn);
         jimmy.start();
         // Now get an MIS of the graph
         iTime.start();
         IntVector_d *fineMIS = AggMIS::MIS::RandomizedMIS(fineDepth, fineGraph);
         iTime.stop();
         // Aggregate to nearest
         iTime.start();
         IntVector_d *fineAgg = AggMIS::Aggregation::AggregateToNearest(fineGraph, *fineMIS);
         iTime.stop();

         // Getting a conditioner
         iTime.start();
         AggMIS::MergeSplitGPU::MergeSplitConditionerGPU fineConditioner(fineGraph, *fineAgg);
         int desiredSize = (fineMin + fineMax) / 2;
         fineConditioner.SetSizeBounds(fineMin, fineMax);
         fineConditioner.Condition(desiredSize, true, .1, .1, 10);
         iTime.stop();
         jimmy.stop();

         // Getting the count of the MIS
         int misCount = thrust::count(fineMIS->begin(), fineMIS->end(), 1);
         //         DataRecorder::Add("Fine MIS Count", misCount);
         fineGraph.indices->swap(adjIndexesIn);
         fineGraph.adjacency->swap(adjacencyIn);
         fineAgg->swap(fineAggregate);

         // Record initial aggregation stats
         Help::RecordAllStats(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               "Initial Fine Aggregation");
         fineAgg->swap(fineAggregate);

         // Swap out the aggregation and graph
         fineConditioner.GetAggregation()->swap(fineAggregate);

         // Record final aggregation stats
         Help::RecordAllStats(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               "Fine Aggregation");

         // Clear temp stuff
         fineMIS->clear();
         delete fineMIS;
         fineAgg->clear();
         delete fineAgg;

         // Setting the permutation array to have values equal to element indices
         permutation = IdxVector_d(numNodesIn);
         misHelpers::fillWithIndex(permutation);

         // Sorting arrays together:
         fineAggregateSort = fineAggregate;
         thrust::sort_by_key(fineAggregateSort.begin(),
               fineAggregateSort.end(),
               permutation.begin());

         // Building the permutation array:
         misHelpers::getInversePermutation(permutation, ipermutation);

         // Getting the aggregate indices and node weights for the induced graph
         misHelpers::getPartSizes(fineAggregateSort,
               inducedNodeWeights,
               aggregateIdx);

         // Getting the induced graph:
         misHelpers::getInducedGraph(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               adjIndexesOut,
               adjacencyOut);

         int inducedGraphSize = adjIndexesOut.size() - 1;

         // Doing the coarse aggregation with AggMIS
         // Swapping in the graph data and weights
         AT::Graph_d coarseGraph;
         coarseGraph.indices->swap(adjIndexesOut);
         coarseGraph.adjacency->swap(adjacencyOut);
         IntVector_d nodeWeights;
         nodeWeights.swap(inducedNodeWeights);

         jimmy.start();
         // Getting an MIS
         iTime.start();
         IntVector_d *coarseMIS = AggMIS::MIS::RandomizedMIS(coarseDepth, coarseGraph);
         iTime.stop();

         // Getting initial aggregation
         iTime.start();
         IntVector_d *coarseAgg = AggMIS::Aggregation::AggregateToNearest(coarseGraph, *coarseMIS);
         iTime.stop();

         // Getting a conditioner
         iTime.start();
         AggMIS::MergeSplitGPU::MergeSplitConditionerGPU coarseConditioner(coarseGraph, *coarseAgg);
         coarseConditioner.SetNodeWeights(nodeWeights);
         coarseConditioner.SetSizeBounds(coarseMin, coarseMax);
         desiredSize = (coarseMin + coarseMax) / 2;
         coarseConditioner.Condition(desiredSize, true, .1, .1, 10);
         jimmy.stop();
         iTime.stop();
         misCount = thrust::count(coarseMIS->begin(), coarseMIS->end(), 1);

         // Swap out the aggregation, graph, and node weights
         coarseGraph.indices->swap(adjIndexesOut);
         coarseGraph.adjacency->swap(adjacencyOut);
         coarseConditioner.GetNodeWeights()->swap(inducedNodeWeights);
         coarseAgg->swap(coarseAggregate);

         // Record initial aggregation stats
         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Initial Coarse Aggregation");
         coarseAgg->swap(coarseAggregate);
         coarseConditioner.GetAggregation()->swap(coarseAggregate);

         // Record final aggregation stats
         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Coarse Aggregation");

         // Clear temp stuff
         coarseMIS->clear();
         delete coarseMIS;
         coarseAgg->clear();
         delete coarseAgg;

         // Performing new version of getting induced graph
         misHelpers::remapInducedGraph(adjIndexesOut, adjacencyOut, coarseAggregate);

         // Filling in the partitionLabel:
         misHelpers::fillPartitionLabel(coarseAggregate, fineAggregateSort, partitionLabel);

         // Do a stable sort by key with the partitionLabel as the key:
         thrust::stable_sort_by_key(partitionLabel.begin(), partitionLabel.end(), thrust::make_zip_iterator(thrust::make_tuple(fineAggregateSort.begin(), permutation.begin())));

         // Remapping the aggregate id's:
         aggregateRemapId = IdxVector_d(aggregateIdx.size() - 1, 0);
         aggregateRemapIndex = IdxVector_d(aggregateIdx.size() - 1, 0);
         misHelpers::fillWithIndex(aggregateRemapId);

         misHelpers::getAggregateStartIndices(fineAggregateSort, aggregateRemapIndex);

         thrust::stable_sort_by_key(aggregateRemapIndex.begin(), aggregateRemapIndex.end(), aggregateRemapId.begin());

         misHelpers::getInversePermutation(aggregateRemapId, iAggregateRemapId);

         misHelpers::remapAggregateIdx(fineAggregateSort, iAggregateRemapId);

         misHelpers::remapAggregateIdx(fineAggregate, iAggregateRemapId);

         // Sort the coarseAggregate for indices and permutation:
         thrust::sort(coarseAggregate.begin(), coarseAggregate.end());

         misHelpers::getPartIndices(coarseAggregate, partitionIdx);

         // Get indices for the fine aggregates
         misHelpers::getPartIndices(fineAggregateSort, aggregateIdx);

         // Putting in the right permutation vectors for the output:
         ipermutation = permutation;
         misHelpers::getInversePermutation(ipermutation, permutation);
      }
      void NewMIS_CPU(IdxVector_d &adjIndexesIn,
            IdxVector_d &adjacencyIn,
            IdxVector_d &permutation,
            IdxVector_d &ipermutation,
            IdxVector_d &aggregateIdx,
            IdxVector_d &partitionIdx,
            IdxVector_d &partitionLabel,
            IdxVector_d &adjIndexesOut,
            IdxVector_d &adjacencyOut,
            int parameters,
            int part_max_size,
            bool verbose) {
         int numNodesIn = adjIndexesIn.size() - 1; // Size of input graph
         IdxVector_d fineAggregate(numNodesIn, 0); // The partition label for the fine partition
         IdxVector_d fineAggregateSort; // The copy of the fine partition label that is sorted
         IdxVector_d finePartSizes; // Vector with sizes of fine partitions
         IdxVector_d neighborCountsIn; // Vector to hold the sizes for each nodes adjacency
         IdxVector_d permutedAdjIndexesIn; // Vector to hold the indices for the initial adjacency permutation
         IdxVector_d permutedAdjacencyIn; // Holds the permuted initial adjacency
         IdxVector_d neighborCountsOut; // Holds the counts of neighbors for the induced graph
         IdxVector_d coarseAggregate; // Holds the partition label for the coarse partition
         IdxVector_d aggregateRemapId; // Holds the current id of each aggregate
         IdxVector_d iAggregateRemapId; // Holds the permutation to remap the aggregate id'
         IdxVector_d aggregateRemapIndex; // Holds the start index of each aggregate
         IdxVector_d inducedNodeWeights; // Holds the sizes of the fine aggregates

         // Creating timer
         AggMIS::Types::JTimer jimmy;
         AggMIS::Types::JTimer iTime;

         // Parsing the parameters
         int fineMin, fineMax, coarseMin, coarseMax, fineDepth, coarseDepth;
         fineMax = parameters % 1000;
         fineMin = (parameters / 1000) % 1000;
         coarseMax = part_max_size % 1000;
         coarseMin = (part_max_size / 1000) % 1000;
         coarseDepth = (parameters / 1000000) % 10;
         fineDepth = (parameters / 10000000) % 10;

         // Initialize the partitionLabel array
         partitionLabel = IdxVector_d(numNodesIn, 2); // Holds the partition each vertex is located in

         // Getting aggregation of graph with AggMIS
         AT::Graph_d fineGraph;
         fineGraph.indices->swap(adjIndexesIn);
         fineGraph.adjacency->swap(adjacencyIn);

         // Getting a host version of the graph
         AT::Graph_h fineGraph_h(fineGraph);
         jimmy.start();

         // Now get an MIS of the graph
         iTime.start();
         AT::IntVector_h *fineMIS = AggMIS::MIS::FloodFillMIS(fineDepth, fineGraph_h);
         iTime.stop();

         // Aggregate to nearest
         iTime.start();
         AT::IntVector_h *fineAgg = AggMIS::Aggregation::AggregateToNearest(fineGraph_h, *fineMIS);
         iTime.stop();

         // Getting a conditioner
         iTime.start();
         AggMIS::MergeSplitCPU::MergeSplitConditionerCPU fineConditioner(fineGraph_h, *fineAgg);

         int desiredSize = (fineMin + fineMax) / 2;
         fineConditioner.SetSizeBounds(fineMin, fineMax);
         fineConditioner.Condition(desiredSize, true, .1, .1, 10);
         jimmy.stop();
         iTime.stop();

         // Getting the count of the MIS
         int misCount = thrust::count(fineMIS->begin(), fineMIS->end(), 1);
         //         DataRecorder::Add("Fine MIS Count", misCount);

         // Swap out the aggregation and graph
         fineGraph.indices->swap(adjIndexesIn);
         fineGraph.adjacency->swap(adjacencyIn);
         fineAggregate.assign(fineAgg->begin(), fineAgg->end());
         Help::RecordAllStats(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               "Initial Fine Aggregation");
         fineAggregate.assign(fineConditioner.GetAggregation()->begin(),
               fineConditioner.GetAggregation()->end());
         Help::RecordAllStats(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               "Fine Aggregation");

         // Clear temp stuff
         fineMIS->clear();
         delete fineMIS;
         fineAgg->clear();
         delete fineAgg;

         // Setting the permutation array to have values equal to element indices
         permutation = IdxVector_d(numNodesIn);
         misHelpers::fillWithIndex(permutation);

         // Sorting arrays together:
         fineAggregateSort = fineAggregate;
         thrust::sort_by_key(fineAggregateSort.begin(),
               fineAggregateSort.end(),
               permutation.begin());

         // Building the permutation array:
         misHelpers::getInversePermutation(permutation, ipermutation);

         // Getting the aggregate indices and node weights for the induced graph
         misHelpers::getPartSizes(fineAggregateSort,
               inducedNodeWeights,
               aggregateIdx);

         // Getting the induced graph:
         misHelpers::getInducedGraph(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               adjIndexesOut,
               adjacencyOut);

         int inducedGraphSize = adjIndexesOut.size() - 1;

         // Doing the coarse aggregation with AggMIS
         // Swapping in the graph data and weights
         AT::Graph_d coarseGraph;
         coarseGraph.indices->swap(adjIndexesOut);
         coarseGraph.adjacency->swap(adjacencyOut);

         AT::Graph_h coarseGraph_h(coarseGraph);
         AT::IntVector_h nodeWeights_h(inducedNodeWeights.begin(),
               inducedNodeWeights.end());
         jimmy.start();

         // Getting an MIS
         iTime.start();
         AT::IntVector_h *coarseMIS = AggMIS::MIS::FloodFillMIS(coarseDepth, coarseGraph_h);
         iTime.stop();

         // Getting initial aggregation
         iTime.start();
         AT::IntVector_h *coarseAgg = AggMIS::Aggregation::AggregateToNearest(coarseGraph_h, *coarseMIS);
         iTime.stop();

         // Getting a conditioner
         iTime.start();
         AggMIS::MergeSplitCPU::MergeSplitConditionerCPU coarseConditioner(coarseGraph_h, *coarseAgg);
         coarseConditioner.SetNodeWeights(nodeWeights_h);
         coarseConditioner.SetSizeBounds(coarseMin, coarseMax);
         desiredSize = (coarseMin + coarseMax) / 2;
         coarseConditioner.Condition(desiredSize, true, .1, .1, 10);
         jimmy.stop();
         iTime.stop();
         misCount = thrust::count(coarseMIS->begin(), coarseMIS->end(), 1);

         // Swap out the aggregation, graph, and node weights
         coarseGraph.indices->swap(adjIndexesOut);
         coarseGraph.adjacency->swap(adjacencyOut);
         coarseAggregate.assign(coarseAgg->begin(), coarseAgg->end());
         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Initial Coarse Aggregation");
         coarseAggregate.assign(coarseConditioner.GetAggregation()->begin(),
               coarseConditioner.GetAggregation()->end());
         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Coarse Aggregation");

         // Clear temp stuff
         coarseMIS->clear();
         delete coarseMIS;
         coarseAgg->clear();
         delete coarseAgg;

         // Performing new version of getting induced graph
         misHelpers::remapInducedGraph(adjIndexesOut, adjacencyOut, coarseAggregate);

         // Filling in the partitionLabel:
         misHelpers::fillPartitionLabel(coarseAggregate, fineAggregateSort, partitionLabel);

         // Do a stable sort by key with the partitionLabel as the key:
         thrust::stable_sort_by_key(partitionLabel.begin(), partitionLabel.end(), thrust::make_zip_iterator(thrust::make_tuple(fineAggregateSort.begin(), permutation.begin())));

         // Remapping the aggregate id's:
         aggregateRemapId = IdxVector_d(aggregateIdx.size() - 1, 0);
         aggregateRemapIndex = IdxVector_d(aggregateIdx.size() - 1, 0);
         misHelpers::fillWithIndex(aggregateRemapId);
         misHelpers::getAggregateStartIndices(fineAggregateSort, aggregateRemapIndex);
         thrust::stable_sort_by_key(aggregateRemapIndex.begin(), aggregateRemapIndex.end(), aggregateRemapId.begin());
         misHelpers::getInversePermutation(aggregateRemapId, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregateSort, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregate, iAggregateRemapId);

         // Sort the coarseAggregate for indices and permutation:
         thrust::sort(coarseAggregate.begin(), coarseAggregate.end());
         misHelpers::getPartIndices(coarseAggregate, partitionIdx);

         // Get indices for the fine aggregates
         misHelpers::getPartIndices(fineAggregateSort, aggregateIdx);

         // Putting in the right permutation vectors for the output:
         ipermutation = permutation;
         misHelpers::getInversePermutation(ipermutation, permutation);
         if (verbose)
            printf("Total aggregation time (Conditioned MIS CPU): %3.4fs for %d calls\n", totalAggregationTime, totalAggregationCalls);
      }
      void LightMIS_CPU(IdxVector_d &adjIndexesIn,
            IdxVector_d &adjacencyIn,
            IdxVector_d &permutation,
            IdxVector_d &ipermutation,
            IdxVector_d &aggregateIdx,
            IdxVector_d &partitionIdx,
            IdxVector_d &partitionLabel,
            IdxVector_d &adjIndexesOut,
            IdxVector_d &adjacencyOut,
            int parameters,
            int part_max_size,
            bool verbose) {
         int numNodesIn = adjIndexesIn.size() - 1; // Size of input graph
         IdxVector_d fineAggregate(numNodesIn, 0); // The partition label for the fine partition
         IdxVector_d fineAggregateSort; // The copy of the fine partition label that is sorted
         IdxVector_d finePartSizes; // Vector with sizes of fine partitions
         IdxVector_d neighborCountsIn; // Vector to hold the sizes for each nodes adjacency
         IdxVector_d permutedAdjIndexesIn; // Vector to hold the indices for the initial adjacency permutation
         IdxVector_d permutedAdjacencyIn; // Holds the permuted initial adjacency
         IdxVector_d neighborCountsOut; // Holds the counts of neighbors for the induced graph
         IdxVector_d coarseAggregate; // Holds the partition label for the coarse partition
         IdxVector_d aggregateRemapId; // Holds the current id of each aggregate
         IdxVector_d iAggregateRemapId; // Holds the permutation to remap the aggregate id'
         IdxVector_d aggregateRemapIndex; // Holds the start index of each aggregate
         IdxVector_d inducedNodeWeights; // Holds the sizes of the fine aggregates

         // Creating timer
         AggMIS::Types::JTimer jimmy;
         AggMIS::Types::JTimer iTime;

         // Parsing the parameters
         int fineDepth, coarseDepth;
         int maxPart = parameters % 1000;
         coarseDepth = (parameters / 1000000) % 10;
         fineDepth = (parameters / 10000000) % 10;


         // Initialize the partitionLabel array
         partitionLabel = IdxVector_d(numNodesIn, 2); // Holds the partition each vertex is located in

         // Getting aggregation of graph with AggMIS
         AT::Graph_d fineGraph;
         fineGraph.indices->swap(adjIndexesIn);
         fineGraph.adjacency->swap(adjacencyIn);

         // Getting a host version of the graph
         AT::Graph_h fineGraph_h(fineGraph);
         jimmy.start();

         // Now get an MIS of the graph
         iTime.start();
         AT::IntVector_h *fineMIS = AggMIS::MIS::FloodFillMIS(fineDepth, fineGraph_h);
         iTime.stop();

         // Aggregate to nearest
         iTime.start();
         AT::IntVector_h *fineAgg = AggMIS::Aggregation::AggregateToNearest(fineGraph_h, *fineMIS);
         iTime.stop();

         // Getting a conditioner
         iTime.start();
         AggMIS::MergeSplitCPU::MergeSplitConditionerCPU fineConditioner(fineGraph_h, *fineAgg);

         jimmy.stop();
         iTime.stop();

         // Getting the count of the MIS
         int misCount = thrust::count(fineMIS->begin(), fineMIS->end(), 1);

         // Swap out the aggregation and graph
         fineGraph.indices->swap(adjIndexesIn);
         fineGraph.adjacency->swap(adjacencyIn);
         fineAggregate.assign(fineAgg->begin(), fineAgg->end());
         Help::RecordAllStats(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               "Initial Fine Aggregation");
         fineAggregate.assign(fineConditioner.GetAggregation()->begin(),
               fineConditioner.GetAggregation()->end());
         Help::RecordAllStats(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               "Fine Aggregation");

         // Clear temp stuff
         fineMIS->clear();
         delete fineMIS;
         fineAgg->clear();
         delete fineAgg;

         // Setting the permutation array to have values equal to element indices
         permutation = IdxVector_d(numNodesIn);
         misHelpers::fillWithIndex(permutation);

         // Sorting arrays together:
         fineAggregateSort = fineAggregate;
         thrust::sort_by_key(fineAggregateSort.begin(),
               fineAggregateSort.end(),
               permutation.begin());

         // Building the permutation array:
         misHelpers::getInversePermutation(permutation, ipermutation);

         // Getting the aggregate indices and node weights for the induced graph
         //            finePartCount = fineAggregateSort[fineAggregateSort.size() - 1];
         misHelpers::getPartSizes(fineAggregateSort,
               inducedNodeWeights,
               aggregateIdx);

         // Getting the induced graph:
         misHelpers::getInducedGraph(adjIndexesIn,
               adjacencyIn,
               fineAggregate,
               adjIndexesOut,
               adjacencyOut);

         int inducedGraphSize = adjIndexesOut.size() - 1;

         // Doing the coarse aggregation with AggMIS
         // Swapping in the graph data and weights
         AT::Graph_d coarseGraph;
         coarseGraph.indices->swap(adjIndexesOut);
         coarseGraph.adjacency->swap(adjacencyOut);

         AT::Graph_h coarseGraph_h(coarseGraph);
         AT::IntVector_h nodeWeights_h(inducedNodeWeights.begin(),
               inducedNodeWeights.end());
         jimmy.start();

         // Getting an MIS
         iTime.start();
         AT::IntVector_h *coarseMIS = AggMIS::MIS::FloodFillMIS(coarseDepth, coarseGraph_h);
         iTime.stop();

         // Getting initial aggregation
         iTime.start();
         AT::IntVector_h *coarseAgg = AggMIS::Aggregation::AggregateToNearest(coarseGraph_h, *coarseMIS);
         iTime.stop();

         // Getting a conditioner
         iTime.start();
         AggMIS::MergeSplitCPU::MergeSplitConditionerCPU coarseConditioner(coarseGraph_h, *coarseAgg);
         coarseConditioner.SetNodeWeights(nodeWeights_h);
         coarseConditioner.SetSizeBounds(maxPart / 2, maxPart);
         coarseConditioner.CycleSplits(true);
         coarseConditioner.CycleMerges(false);
         jimmy.stop();
         iTime.stop();
         misCount = thrust::count(coarseMIS->begin(), coarseMIS->end(), 1);

         // Swap out the aggregation, graph, and node weights
         coarseGraph.indices->swap(adjIndexesOut);
         coarseGraph.adjacency->swap(adjacencyOut);
         coarseAggregate.assign(coarseAgg->begin(), coarseAgg->end());
         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Initial Coarse Aggregation");
         coarseAggregate.assign(coarseConditioner.GetAggregation()->begin(),
               coarseConditioner.GetAggregation()->end());
         Help::RecordAllStats(adjIndexesOut,
               adjacencyOut,
               coarseAggregate,
               inducedNodeWeights,
               "Coarse Aggregation");

         // Clear temp stuff
         coarseMIS->clear();
         delete coarseMIS;
         coarseAgg->clear();
         delete coarseAgg;

         // Performing new version of getting induced graph
         misHelpers::remapInducedGraph(adjIndexesOut, adjacencyOut, coarseAggregate);

         // Filling in the partitionLabel:
         misHelpers::fillPartitionLabel(coarseAggregate,
               fineAggregateSort,
               partitionLabel);

         // Do a stable sort by key with the partitionLabel as the key:
         thrust::stable_sort_by_key(partitionLabel.begin(),
               partitionLabel.end(),
               thrust::make_zip_iterator(
                  thrust::make_tuple(
                     fineAggregateSort.begin(),
                     permutation.begin())));

         // Remapping the aggregate id's:
         aggregateRemapId = IdxVector_d(aggregateIdx.size() - 1, 0);
         aggregateRemapIndex = IdxVector_d(aggregateIdx.size() - 1, 0);
         misHelpers::fillWithIndex(aggregateRemapId);
         misHelpers::getAggregateStartIndices(fineAggregateSort, aggregateRemapIndex);
         thrust::stable_sort_by_key(aggregateRemapIndex.begin(), aggregateRemapIndex.end(), aggregateRemapId.begin());
         misHelpers::getInversePermutation(aggregateRemapId, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregateSort, iAggregateRemapId);
         misHelpers::remapAggregateIdx(fineAggregate, iAggregateRemapId);

         // Sort the coarseAggregate for indices and permutation:
         thrust::sort(coarseAggregate.begin(), coarseAggregate.end());
         misHelpers::getPartIndices(coarseAggregate, partitionIdx);

         // Get indices for the fine aggregates
         misHelpers::getPartIndices(fineAggregateSort, aggregateIdx);

         // Putting in the right permutation vectors for the output:
         ipermutation = permutation;
         misHelpers::getInversePermutation(ipermutation, permutation);
      }
   }
   namespace Help {
      namespace AT = AggMIS::Types;
      int GetMetisAggregation(AT::IntVector_h &indices,
            AT::IntVector_h &adjacency,
            AT::IntVector_h &result,
            int partSize,
            bool verbose) {
         // Getting size of graph
         int graphSize = indices.size() - 1;

         // Making sure result is sized correctly
         result.resize(graphSize);

         // Setting up for Metis call:
         int nparts, edgecut;
         int *npart = &result[0];
         nparts = (graphSize / partSize);
         if (nparts < 8192) {
            if (nparts < 2)
               nparts = 2;
            int options[10], pnumflag = 0, wgtflag = 0;
            for (int i = 0; i < 10; i++)
               options[i] = 0;

            AggMIS::Types::JTimer jimmy;
            jimmy.start();

            METIS_PartGraphKway(&graphSize,
                  &indices[0],
                  &adjacency[0],
                  NULL,
                  NULL,
                  &wgtflag,
                  &pnumflag,
                  &nparts,
                  options,
                  &edgecut,
                  npart);
            jimmy.stop();
            if (verbose)
               printf("Metis call for graph of %d nodes into parts of size %d took %3.4fs (host) %3.4fs (cuda)\n", graphSize, partSize, jimmy.getElapsedTimeInSec(true), jimmy.getElapsedTimeInSec(false));

            totalAggregationTime += jimmy.getElapsedTimeInSec(true);
            totalAggregationCalls++;

            // Output timing to file
            std::ofstream outputFile;
            outputFile.open("TimingResults.csv", std::ofstream::app);
            if (totalAggregationCalls == 1)
               outputFile << "\n\nNote,Metis Aggregation Time, Total Calls, Total Time\n";
            outputFile << "Metis call on graph with " << graphSize << " nodes into parts of size " << partSize << ",";
            outputFile << jimmy.getElapsedTimeInSec(true) <<
               "," << totalAggregationCalls << "," << totalAggregationTime << "\n";

            return EnsureConnectedAndNonEmpty(indices, adjacency, result);
         }
         else {
            int count = GetMetisAggregation_Large(indices, adjacency, result, partSize);
            return count;
         }
      }
      int GetMetisAggregation_Large(AT::IntVector_h &indices,
            AT::IntVector_h &adjacency,
            AT::IntVector_h &result,
            int partSize,
            bool verbose) {
         // Getting size of graph
         int graphSize = indices.size() - 1;

         // Getting a partitioning with four parts to create subgraphs
         int subGraphSize = graphSize / 4;
         GetMetisAggregation(indices, adjacency, result, subGraphSize);

         // Getting the subgraphs from the partitioning
         AT::IntVector_h_ptr subIndices, subAdjacencies, subNodeMaps;
         GetSubGraphs(indices, adjacency, result, subIndices, subAdjacencies, subNodeMaps);
         int subGraphCount = subIndices.size();

         // Getting aggregation of each subgraph and mapping to original graph
         int offset = 0;
         for (int i = 0; i < subGraphCount; i++) {
            AT::IntVector_h agg;
            AT::IntVector_h &nodeMap = *(subNodeMaps[i]);
            AT::IntVector_h &ind = *(subIndices[i]);
            AT::IntVector_h &adj = *(subAdjacencies[i]);

            int aggCount = GetMetisAggregation(ind, adj, agg, partSize);
            for (int n = 0; n < agg.size(); n++) {
               // Look up original node Id
               int node = nodeMap[n];
               // Set the aggregate to the subgraph's plus current offset
               result[node] = agg[n] + offset;
            }
            offset += aggCount;
            agg.clear();
            ind.clear();
            adj.clear();
            nodeMap.clear();
         }

         // Cleaning up
         subIndices.clear();
         subAdjacencies.clear();
         subNodeMaps.clear();

         // Return number of aggregates
         return offset;
      }
      void GetSubGraphs(AT::IntVector_h &indices,
            AT::IntVector_h &adjacency,
            AT::IntVector_h &partition,
            AT::IntVector_h_ptr &newIndices,
            AT::IntVector_h_ptr &newAdjacencies,
            AT::IntVector_h_ptr &nodeMaps,
            bool verbose) {

         // Getting a map from old graph id to subgraph id
         AT::IntVector_h mapToSubGraphId(adjacency.size() - 1);

         // Getting separate node maps for all partitions
         nodeMaps.clear();
         int minPart = partition[0];
         int maxPart = partition[0];
         for (int i = 0; i < partition.size(); i++) {
            int partId = partition[i];
            minPart = std::min(minPart, partId);
            maxPart = std::max(maxPart, partId);
            while (partId + 1 > nodeMaps.size())
               nodeMaps.push_back(new AT::IntVector_h());
            nodeMaps[partId]->push_back(i);
            mapToSubGraphId[i] = nodeMaps[partId]->size() - 1;
         }
         int graphCount = nodeMaps.size();

         // Creating the new subgraph indices and adjacency vectors
         newIndices.resize(graphCount);
         newAdjacencies.resize(graphCount);
         for (int i = 0; i < graphCount; i++) {
            newIndices[i] = new AT::IntVector_h(nodeMaps[i]->size() + 1);

            AT::IntVector_h *ptr = newIndices[i];

            newAdjacencies[i] = new AT::IntVector_h();
         }

         // Filling the subgraphs in
         for (int i = 0; i < graphCount; i++) {
            AT::IntVector_h &nodes = *nodeMaps[i];
            AT::IntVector_h &ind = *newIndices[i];
            AT::IntVector_h &adj = *newAdjacencies[i];
            int insertAt = 0;
            (*newIndices[0])[0] = 0;
            for (int nIt = 0; nIt < nodes.size(); nIt++) {
               int node = nodes[nIt];

               if (partition[node] != i) {
                  int p = partition[node];
                  if (verbose)
                     printf("Node %d found in node list %d but marked as in partition %d\n",
                           node, i, p);
                  std::cin >> p;
               }

               int start = indices[node];
               int end = indices[node + 1];
               for (int n = start; n < end; n++) {
                  int neighbor = adjacency[n];
                  if (partition[neighbor] == i) {
                     newAdjacencies[i]->push_back(mapToSubGraphId[neighbor]);
                     insertAt++;
                  }
               }
               ind[nIt + 1] = insertAt;
            }
         }

         // Cleaning up
         mapToSubGraphId.clear();
      }
      int EnsureConnectedAndNonEmpty(AT::IntVector_h &indices,
            AT::IntVector_h &adjacency,
            AT::IntVector_h &aggregation) {
         AT::IntVector_h temp(aggregation.size());

         // Flood fill aggregates with node indices
         for (int i = 0; i < temp.size(); i++)
            temp[i] = i;
         bool changed = true;
         while (changed) {
            changed = false;
            for (int root = 0; root < aggregation.size(); root++) {
               int rootValue = temp[root];
               int rootAggregate = aggregation[root];
               int start = indices[root];
               int end = indices[root + 1];
               for (int nIt = start; nIt < end; nIt++) {
                  int neighbor = adjacency[nIt];
                  int neighborAggregate = aggregation[neighbor];
                  int neighborValue = temp[neighbor];
                  if (rootAggregate == neighborAggregate && neighborValue > rootValue)
                     rootValue = neighborValue;
               }
               if (rootValue > temp[root]) {
                  temp[root] = rootValue;
                  changed = true;
               }
            }
         }

         // Making a copy of the filled aggregation
         AT::IntVector_h mapping(temp.size());
         thrust::copy(temp.begin(), temp.end(), mapping.begin());


         // Sort the values
         thrust::sort(mapping.begin(), mapping.end());

         // Get just unique values
         int newSize = thrust::unique(mapping.begin(), mapping.end()) - mapping.begin();
         mapping.resize(newSize);

         // Remap aggregation
         for (int i = 0; i < aggregation.size(); i++)
            aggregation[i] = BinarySearch(temp[i], mapping);

         // Get rid of temporary vectors
         mapping.clear();
         temp.clear();

         // Return count of aggregates
         return newSize;
      }
      int BinarySearch(int value,
            AT::IntVector_h &array) {
         int imin = 0;
         int imax = array.size() - 1;
         while (imin < imax) {
            int imid = (imax + imin) / 2;
            if (array[imid] < value)
               imin = imid + 1;
            else
               imax = imid;
         }
         if (imax == imin && array[imin] == value)
            return imin;
         else
            return -1;
      }
      void RecordAllStats(IdxVector_d& indices,
            IdxVector_d& adjacency,
            IdxVector_d& aggregation,
            std::string prefix) {
         IdxVector_d dummy;
         RecordAllStats(indices,
               adjacency,
               aggregation,
               dummy,
               prefix);
      }
      void RecordAllStats(IdxVector_d& indices,
            IdxVector_d& adjacency,
            IdxVector_d& aggregation,
            IdxVector_d& nodeWeights,
            std::string prefix) {
         // Recording aggregation stats
         if (nodeWeights.size() == 0)
            RecordAggregationStats(aggregation, prefix + ":Parts");
         else
            RecordAggregationStats(aggregation, nodeWeights, prefix + ":Parts");

         // Recording Valence stats
         RecordValenceStats(indices, adjacency, prefix + ":Valence");

         // Recording Edge cut ratio
         RecordEdgeCut(indices, adjacency, aggregation, prefix);
      }
      void RecordAggregationStats(IdxVector_d& aggregation,
            std::string prefix) {
         AT::IntVector_d agg;
         agg.swap(aggregation);
         AT::IntVector_d partSizes;

         // Get the part sizes
         AggMIS::GraphHelpers::getPartSizes(agg, partSizes);

         // Find the largest and smallest parts
         thrust::sort(partSizes.begin(), partSizes.end());
         int smallest = partSizes[0];
         int largest = partSizes.back();

         // Get the mean, median, and std deviation
         double meanSize = (double)agg.size() / partSizes.size();
         int medianSize = partSizes[(partSizes.size() - 1) / 2];
         double std = thrust::transform_reduce(partSizes.begin(),
               partSizes.end(),
               AggMIS::MergeSplitGPU::Functors::SquaredDifference(meanSize),
               0.0,
               thrust::plus<double>());
         std = sqrt(std / partSizes.size());

         agg.swap(aggregation);
      }
      void RecordAggregationStats(IdxVector_d& aggregation,
            IdxVector_d& nodeWeights,
            std::string prefix) {
         AT::IntVector_d agg;
         agg.swap(aggregation);
         AT::IntVector_d nw;
         nw.swap(nodeWeights);
         AT::IntVector_d partSizes;

         // Get the part sizes
         AggMIS::GraphHelpers::getPartSizes(agg, partSizes, nw);


         // Find the largest and smallest parts
         thrust::sort(partSizes.begin(), partSizes.end());
         int smallest = partSizes[0];
         int largest = partSizes.back();

         // Get the mean, median, and std deviation
         int totalWeight = thrust::reduce(nw.begin(), nw.end());
         int medianSize = partSizes[(partSizes.size() - 1) / 2];
         double meanSize = (double)totalWeight / partSizes.size();
         double std = thrust::transform_reduce(partSizes.begin(),
               partSizes.end(),
               AggMIS::MergeSplitGPU::Functors::SquaredDifference(meanSize),
               0.0,
               thrust::plus<double>());
         std = sqrt(std / partSizes.size());

         agg.swap(aggregation);
         nw.swap(nodeWeights);
      }
      void RecordValenceStats(IdxVector_d& indices,
            IdxVector_d& adjacency,
            std::string prefix) {
         // Get a graph object to use
         AT::Graph_d g;
         g.adjacency->swap(adjacency);
         g.indices->swap(indices);

         // Get the valences from the graph
         AT::IntVector_d* valences = AggMIS::GraphHelpers::GetValences(g);

         // Compute the stats
         thrust::sort(valences->begin(), valences->end());
         int smallest = valences->data()[0];
         int largest = valences->back();

         // Get the mean, median, and std deviation
         int totalValence = thrust::reduce(valences->begin(),
               valences->end());
         double meanSize = (double)totalValence / valences->size();
         int medianSize = valences->data()[(valences->size() - 1) / 2];
         double std = thrust::transform_reduce(valences->begin(),
               valences->end(),
               AggMIS::MergeSplitGPU::Functors::SquaredDifference(meanSize),
               0.0,
               thrust::plus<double>());
         std = sqrt(std / valences->size());

         g.adjacency->swap(adjacency);
         g.indices->swap(indices);
      }
      void RecordEdgeCut(IdxVector_d& indices,
            IdxVector_d& adjacency,
            IdxVector_d& aggregation,
            std::string prefix) {
         // Get a graph
         AT::Graph_d g;
         g.indices->swap(indices);
         g.adjacency->swap(adjacency);

         // Get an IntVector for aggregation
         AT::IntVector_d agg;
         agg.swap(aggregation);

         // Swapping back data
         g.indices->swap(indices);
         g.adjacency->swap(adjacency);
         agg.swap(aggregation);
      }
   }
}
